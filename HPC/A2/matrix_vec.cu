#include<bits/stdc++.h>
#include<hip/hip_runtime.h>

using namespace std;

#define N 2048


void initialise(int* v,int n){
  for(int i = 0;i<n;i++){
    v[i] = rand()%1000;
  }
}

__global__ void mat_vec_mult(int* mat,int*v,int* res,int n){
  int tid = threadIdx.x;
  res[tid] = 0;
  for(int i =0;i<n;i++){
    res[tid] = res[tid] + mat[ tid*n + i]*v[i];
  }
}


int main(){

  int *h_mat,*h_v,*h_r;
  int *d_mat,*d_v,*d_r;
  int *s_r;

  float elapsed_cpu, elapsed_gpu;
  clock_t t1, t2;

  h_mat =(int*)malloc(N*N*sizeof(int));
  h_v = (int*)malloc(N*sizeof(int));
  h_r = (int*)malloc(N*sizeof(int));
  s_r = (int*)malloc(N*sizeof(int));

  hipMalloc(&d_mat,N*N*sizeof(int));
  hipMalloc(&d_v,N*sizeof(int));
  hipMalloc(&d_r,N*sizeof(int));

  initialise(h_mat,N*N);
  initialise(h_v,N);

  //serial
  t1 = clock();
  for(int i =0;i<N;i++){
    s_r[i]=0;
    for(int j=0;j<N;j++){
      s_r[i] = s_r[i] + h_mat[i*N + j]*h_v[j];
    }
  }
  t2 = clock();


  //parallel

  hipEvent_t start, stop;

  hipEventCreate(&start);

  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  hipMemcpy(d_mat,h_mat,N*N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_v,h_v,N*sizeof(int),hipMemcpyHostToDevice);

  mat_vec_mult<<<1,N>>>(d_mat,d_v,d_r,N);

  hipMemcpy(h_r,d_r,N*sizeof(int),hipMemcpyDeviceToHost);


  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsed_gpu, start, stop);

  hipEventDestroy(start);

  hipEventDestroy(stop);

  elapsed_cpu = ((float)t2 - (float)t1) / CLOCKS_PER_SEC * 1000;	//cpu elapsed time in ms

  cout<<elapsed_cpu<<endl;
  cout<<elapsed_gpu<<endl;
  cout<<"speedup "<<elapsed_cpu/elapsed_gpu<<endl;

  for(int i =0;i<N;i++){
    if(s_r[i]!=h_r[i]){
      cout<<"failed";
      break;
    }
  }






  return 0;
}
