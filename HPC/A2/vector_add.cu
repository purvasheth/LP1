
#include <hip/hip_runtime.h>

#include <cstdlib>
#include <stdio.h>
#include <cassert>
#include <iostream>

// CUDA kernel for vector addition
__global__ void vectorAdd(int* a, int* b, int* c, int n) {
	// Calculate global thread ID (tid)
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	// Vector boundary guard
	if (tid < n) {
		// Each thread adds a single element
		c[tid] = a[tid] + b[tid];
	}
}

// Initialize vector of size n to int between 0-99
void vector_init(int* a, int n) {
	for (int i = 0; i < n; i++) {
		a[i] = rand() % 100;
	}
}

// Check vector add result
void check_answer(int* a, int* b, int*c, int n) {
	for (int i = 0; i < n; i++) {
		assert(c[i] == a[i] + b[i]);
	}
}

int main() {
	// Vector size of 2^16 (65536 elements)
	int n = 1 << 16;
	// Host vector pointers
	int *h_a, *h_b, *h_c;
	// Device vector pointers
	int *d_a, *d_b, *d_c;
	// Allocation size for all vectors
	size_t bytes = sizeof(int) * n;

	// Allocate host memory
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	// Allocate device memory
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Initialize vectors a and b with random values between 0 and 99
	vector_init(h_a, n);
	vector_init(h_b, n);

	// Copy data from
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// Threadblock size
	int NUM_THREADS = 256;

	// Grid size
	int NUM_BLOCKS = (n) / NUM_THREADS;

	// Launch kernel on default stream w/o shmem
	vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, n);

	// Copy sum vector from device to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Check result for errors
	check_answer(h_a, h_b, h_c, n);

    // Free memory on host
    free(h_a);
    free(h_b);
    free(h_c);

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	printf("COMPLETED SUCCESFULLY\n");

	return 0;
}
