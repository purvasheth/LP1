#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

__global__ void matrixMul(int *a, int *b, int *c, int n) {
	// Compute each thread's row
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	// Compute each thread's column
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int temp_sum = 0;
	// Boundary protection
	if ((row < n) && (col < n)) {
		// Iterate over row, and down column
		for (int k = 0; k < n; k++) {
			// Accumulate result for a single element
			temp_sum += a[row * n + k] * b[k * n + col];
		}
		// Assign result
		c[row * n + col] = temp_sum;
	}

}

// Initialization function for matrices
void matrix_init(int *a, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			a[i * n + j] = rand() % 100;
		}
	}
}

// Check result
void check_answer(int *a, int *b, int *c, int n) {
	int *verify_c;
	verify_c = (int*)malloc(n * n * sizeof(int));
	int temp_sum;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			temp_sum = 0;
			for (int k = 0; k < n; k++) {
				temp_sum += a[i * n + k] * b[k * n + j];
			}
			verify_c[i * n + j] = temp_sum;
		}
	}

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			assert(c[i * n + j] == verify_c[i * n + j]);
		}
	}
}

int main() {
	// Matrix size of 1024 x 1024;
	int n = 1 << 10;

	// Size (in bytes) of matrix
	size_t bytes = n * n * sizeof(int);

	// Host pointers
	int *h_a, *h_b, *h_c;

	// Allocate host memory
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	// Device pointers
	int *d_a, *d_b, *d_c;

	// Allocated device memory
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Initialize matrices
	matrix_init(h_a, n);
	matrix_init(h_b, n);

	// Copy data to the device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// Threads per block
	int BLOCK_SIZE = 16;

	// Blocks in each dimension (No padding)
	int GRID_SIZE = n / BLOCK_SIZE;

	// Use dim3 objects
	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel
	matrixMul <<<grid, threads >>> (d_a, d_b, d_c, n);

	// Copy back to the host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Check result
	check_answer(h_a, h_b, h_c, n);

    // Free memory on host
    free(h_a);
    free(h_b);
    free(h_c);

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	printf("COMPLETED SUCCESSFULLY\n");

	return 0;
}
