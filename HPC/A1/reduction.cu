#include<bits/stdc++.h>

#include<hip/hip_runtime.h>
using namespace std;

#define SIZE 256
#define SSIZE SIZE*4 // sizeof(int)

__global__ void sum_reduction(int *v,int *v_r){

  __shared__ int partial_sum[SSIZE];

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  partial_sum[threadIdx.x] = v[tid];

  __syncthreads();

  for(int s = blockDim.x/2;s>0;s=s/2){
      if(threadIdx.x < s){
        partial_sum[threadIdx.x] += partial_sum[threadIdx.x+s];
      }
      __syncthreads();
  }
  if(threadIdx.x ==0){
    v_r[blockIdx.x] = partial_sum[0];
  }
}

__global__ void max_reduction(int *v,int *v_r){

  __shared__ int partial_sum[SSIZE];

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  partial_sum[threadIdx.x] = v[tid];

  __syncthreads();

  for(int s = blockDim.x/2;s>0;s=s/2){
      if(threadIdx.x < s){
        partial_sum[threadIdx.x] = max(partial_sum[threadIdx.x],partial_sum[threadIdx.x+s]);
      }
      __syncthreads();
  }
  if(threadIdx.x ==0){
    v_r[blockIdx.x] = partial_sum[0];
  }
}

__global__ void variance(int *v,int *v_r,float *mean){

  __shared__ int partial_sum[SSIZE];

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  partial_sum[threadIdx.x] = v[tid];

  __syncthreads();

  partial_sum[threadIdx.x] = (partial_sum[threadIdx.x] - *mean) * (partial_sum[threadIdx.x] - *mean);

  __syncthreads();

  for(int s = blockDim.x/2;s>0;s=s/2){
      if(threadIdx.x < s){
        partial_sum[threadIdx.x] += partial_sum[threadIdx.x+s];
      }
      __syncthreads();
  }
  if(threadIdx.x ==0){
    v_r[blockIdx.x] = partial_sum[0];
  }
}

__global__ void min_reduction(int *v,int *v_r){

  __shared__ int partial_sum[SSIZE];

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  partial_sum[threadIdx.x] = v[tid];

  __syncthreads();

  for(int s = blockDim.x/2;s>0;s=s/2){
      if(threadIdx.x < s){
        partial_sum[threadIdx.x] = min(partial_sum[threadIdx.x],partial_sum[threadIdx.x+s]);
      }
      __syncthreads();
  }
  if(threadIdx.x ==0){
    v_r[blockIdx.x] = partial_sum[0];
  }
}

void inititialise(int* v,int n){
  for(int i =0;i<n;i++){
    v[i]= rand()%1000;
  }
}


int main(){
  int n = SIZE*SIZE;

  float elapsed_cpu, elapsed_gpu;
  clock_t t1, t2;


  int thread_block_size = SIZE;
  int num_blocks = n / thread_block_size;


  int *h_v,*d_v,*h_v_r,*d_v_r;
  float *d_mean;
  h_v = (int*)malloc(n*sizeof(int));
  hipMalloc(&d_v,n*sizeof(int));
  h_v_r = (int*)malloc(num_blocks*sizeof(int));
  hipMalloc(&d_v_r,num_blocks*sizeof(int));
  hipMalloc((void**)&d_mean,sizeof(float));

  inititialise(h_v,n);

  int minimum = 0;
  for(int i =0;i<n;i++){
    minimum = minimum+h_v[i];
  }
  //cout<<minimum<<endl;

 float mean = minimum / n;
int var = 0;
t1 = clock();
  for(int i =0;i<n;i++){
    var = var + (h_v[i]-mean)*(h_v[i]-mean);
  }
  cout<<var<<endl;
t2 = clock();
elapsed_cpu = ((float)t2 - (float)t1) / CLOCKS_PER_SEC * 1000;	//cpu elapsed time in ms



	hipEvent_t start, stop;

	hipEventCreate(&start);

	hipEventCreate(&stop);

	hipEventRecord(start, 0);

  hipMemcpy(d_v,h_v,n*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_mean,&mean,sizeof(float),hipMemcpyHostToDevice);

  variance<<<num_blocks,thread_block_size>>>(d_v,d_v_r,d_mean);
  sum_reduction<<<1,thread_block_size>>>(d_v_r,d_v_r);

  hipMemcpy(h_v_r,d_v_r,thread_block_size*sizeof(int),hipMemcpyDeviceToHost);
  cout<<h_v_r[0]<<endl;

  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsed_gpu, start, stop);

  hipEventDestroy(start);

  hipEventDestroy(stop);

  cout<<elapsed_cpu<<endl;
  cout<<elapsed_gpu<<endl;
  cout<<"speedup"<<elapsed_cpu/elapsed_gpu<<endl;

  return 0;
}
